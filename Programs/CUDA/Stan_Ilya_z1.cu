
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#define N 4

using namespace std;

/*
Задача:
 Решить СЛАУ (Ax = f, A - матрица N*N) итерационным методом.
 Выбранный метод - Метод простой итерации (Simple iteration method / SIM):  x(k+1) = A'* x(k) + t*f, где
 t = const, 0 < t < 2/||A|| (Нашёл теорему: А=А*>0, 0 < тау < 2/||A|| => SIM сходится).
 A' = E - t*A

 Алгоритм, описанный здесь, сходится не для всех А. Проверок на сходимость нет. Если не сходится, то будет ломаться
или бесконечно считать итерации.

 Глобальные функции-ядра помечены G_.
 У функций, использующих редукцию, написано (reduction)
 (Использовал reduce3)
 У функций, использующих ядра, написано uses G_<название ядра>

 Используемые нормы: для поиска тау - Эйлерова норма, для eps (x(k+1) - x(k)) - L1 норма (сумма модулей).
*/

//_______________________________functions (prototypes)

__global__ void G_alphaMulVec(float *vec, unsigned long int pitch, float mul); // alpha*vector

__global__ void G_vecSubVec(float *resVec, float *vec); // vector-vector

__global__ void G_vecPlusVec(float *resVec, float *vec); // vector+vector

__global__ void G_E_plus_A(float A[N * N], unsigned long int pitch); // E+A

__global__ void G_vecMulVec(float *v1, float *v2, float *res); // vector*vector=res (reduction)

void matrMulVec(float A[N * N], unsigned long int pitch, float vec[N], float *res); // A*vector=res, uses vecMulVec

__global__ void G_absSumOfVecElems(float *vecIn, double *vecOut); // abs(v0)+abs(v1)+abs(v2)+...=res; (reduction)

__global__ void G_squaredEulerNorm(float *In, unsigned long int pitch, float *vecOut); // ||A||^2, Euler (reduction)

__global__ void G_sumOfVecElems(float *vecIn, float *vecOut); // v0+v1+v2+...=res; (reduction)

float coeffSIM(float *A, unsigned long int pitch); // coeff. for simple iter. method = 2/(||A|| + 1), uses G_squaredEulerNorm and G_sumOfVecElems

void coutMatr(float A[N * N]);

//__________________________________________________main()

int main()
{
    // Host init.

    double *eps, H_eps = 1; // eps- on device, H_eps= on Host
    float A[N * N] = {5, 2, 2, 0,
                      3, 10, 4, 0,
                      3, 4, 18, 0,
                      2, 0, 4, 7}; // ||A||^2 = 576 = 24*24 => tau = 0.08 = 2/(24+1)
    // answer: (16/35, 1/5, 11/70, -54/245)
    unsigned long int pitch;
    float f[N] = {3, 4, 5, 0};
    float x0[N] = {1, 0, 1, 2}; // any x0
    float tau;

    // Device init.

    float *D_A, *Df, *Dx0, *Dtmp; // pointers on Device

    hipMallocPitch((void **)&D_A, &pitch, N * sizeof(float), N);
    hipMemcpy2D(D_A, pitch, A, N * sizeof(float), N * sizeof(float), N, hipMemcpyDefault);
    pitch /= sizeof(float); // pitch - ammount of floats in a raw

    hipMalloc((void **)&Df, N * sizeof(float));
    hipMemcpy(Df, f, N * sizeof(float), hipMemcpyDefault);

    hipMalloc((void **)&Dtmp, N * sizeof(float));

    hipMalloc((void **)&Dx0, N * sizeof(float));
    hipMemcpy(Dx0, x0, N * sizeof(float), hipMemcpyDefault);

    hipHostMalloc((void **)&eps, sizeof(double), hipHostMallocDefault);
    // if eps in managed => time=1.7 ms. If eps in host => time=0.48 ms. If eps on device => time=0.53 ms. (1660 ti)

    //  x(k+1) = A'* x(k) + t*f

    tau = coeffSIM(D_A, pitch);

    G_alphaMulVec<<<N, N>>>(D_A, pitch, -tau); // -xA
    G_E_plus_A<<<1, N>>>(D_A, pitch);          // E-xA = E+(-xA) = A'
    G_alphaMulVec<<<1, N>>>(Df, 0, tau);

    // ready to start SIM

    // output of new Matrix А'
    hipMemcpy2D(A, N * sizeof(float), D_A, pitch * sizeof(float), N * sizeof(float), N, hipMemcpyDefault);
    cout << "Matrix A' = E - tA:\n";
    coutMatr(A);

    hipEvent_t Ev1, Ev2; // time measurement
    hipEventCreate(&Ev1);
    hipEventRecord(Ev1, 0);

    // iterations

    while (H_eps > 0.001) // tmp = x(k+1), x0 = x(k)
    {
        matrMulVec(D_A, pitch, Dx0, Dtmp);
        G_vecPlusVec<<<1, N>>>(Dtmp, Df);
        G_vecSubVec<<<1, N>>>(Dx0, Dtmp);

        G_absSumOfVecElems<<<1, N>>>(Dx0, eps); // new epsilon
        hipMemcpy(&H_eps, eps, sizeof(double), hipMemcpyDefault);
        hipMemcpy(Dx0, Dtmp, N * sizeof(float), hipMemcpyDeviceToDevice); // x(k)<->x(k+1)
    }

    hipEventCreate(&Ev2); // time measurement
    hipEventRecord(Ev2, 0);
    hipEventSynchronize(Ev2);
    hipEventElapsedTime(&tau, Ev1, Ev2);
    hipEventDestroy(Ev1);
    hipEventDestroy(Ev2);
    cout << "\ntime: " << tau << endl;

    // result output

    hipMemcpy(x0, Dx0, N * sizeof(float), hipMemcpyDefault);
    cout << "____________\n";
    for (int i = 0; i < N; ++i)
    {
        cout << x0[i] << endl;
    }
    cout << "____________\n";

    // frees

    cout << "CUDA_ERROR: " << hipGetErrorString(hipGetLastError()) << endl;
    hipFree(D_A);
    hipFree(Dx0);
    hipFree(Dtmp);
    hipHostFree(eps);
    hipFree(Df);

    return 0;
}

//___________________________________________functions (definitions)

__global__ void G_alphaMulVec(float *vec, unsigned long int pitch, float mul)
{
    vec[pitch * blockIdx.x + threadIdx.x] *= mul; // Nxblocks, Nxthreads
}

__global__ void G_vecSubVec(float *resVec, float *vec)
{
    resVec[threadIdx.x] -= vec[threadIdx.x];
}

__global__ void G_vecPlusVec(float *resVec, float *vec)
{
    resVec[threadIdx.x] += vec[threadIdx.x];
}

__global__ void G_E_plus_A(float A[N * N], unsigned long int pitch)
{
    unsigned int tid = threadIdx.x;
    ++A[pitch * tid + tid];
}

__global__ void G_vecMulVec(float *v1, float *v2, float *res)
{
    __shared__ float shMass[N];
    unsigned int tid = threadIdx.x;
    shMass[tid] = v1[tid] * v2[tid];
    __syncthreads();
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (tid < s)
        {
            shMass[tid] += shMass[tid + s];
        }
        __syncthreads();
    }
    if (tid == 0)
        *res = shMass[0];
}

void matrMulVec(float A[N * N], unsigned long int pitch, float vec[N], float *res)
{
    for (int i = 0; i < N; i++)
        G_vecMulVec<<<1, N>>>(&A[pitch * i], vec, &res[i]);
}

__global__ void G_absSumOfVecElems(float *vecIn, double *vecOut)
{
    __shared__ float shMass[N];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    shMass[tid] = abs(vecIn[i]);
    __syncthreads();
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (tid < s)
        {
            shMass[tid] += shMass[tid + s];
        }
        __syncthreads();
    }
    if (tid == 0)
        vecOut[blockIdx.x] = shMass[0];
}

__global__ void G_squaredEulerNorm(float *In, unsigned long int pitch, float *vecOut)
{
    __shared__ float shMass[N];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * pitch + threadIdx.x;
    shMass[tid] = In[i] * In[i];
    __syncthreads();
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (tid < s)
        {
            shMass[tid] += shMass[tid + s];
        }
        __syncthreads();
    }
    if (tid == 0)
        vecOut[blockIdx.x] = shMass[0];
}

__global__ void G_sumOfVecElems(float *vecIn, float *vecOut)
{
    __shared__ float shMass[N];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    shMass[tid] = vecIn[i];
    __syncthreads();
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (tid < s)
        {
            shMass[tid] += shMass[tid + s];
        }
        __syncthreads();
    }
    if (tid == 0)
        vecOut[blockIdx.x] = shMass[0];
}

float coeffSIM(float *A, unsigned long int pitch)
{
    float *D_coeff, coeff = 0;
    hipMalloc((void **)&D_coeff, N * sizeof(float));
    G_squaredEulerNorm<<<N, N>>>(A, pitch, D_coeff);
    G_sumOfVecElems<<<1, N>>>(D_coeff, D_coeff);

    hipMemcpy(&coeff, D_coeff, sizeof(float), hipMemcpyDefault);
    hipFree(D_coeff);
    coeff = 2 / (sqrt(coeff) + 1);
    return coeff; // tau < 2/||A|| (if A=A*>0 => simple iter. will converge (theory))
}

void coutMatr(float A[N * N])
{
    for (int j = 0; j < N; j++)
    {
        for (int i = 0; i < N; i++)
        {
            printf("%.3f  ", A[j * N + i]);
        }
        cout << endl;
    }
}
