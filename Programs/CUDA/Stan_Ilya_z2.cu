#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#define SQR(x) ((float)(x) * (float)(x))
#define bufSize 1024 * 1024
#define BLUR_PARAM 40
using namespace std;

// CIEXYZTRIPLE stuff
typedef int FXPT2DOT30;

struct CIEXYZ
{
	FXPT2DOT30 ciexyzX;
	FXPT2DOT30 ciexyzY;
	FXPT2DOT30 ciexyzZ;
};

struct CIEXYZTRIPLE
{
	CIEXYZ ciexyzRed;
	CIEXYZ ciexyzGreen;
	CIEXYZ ciexyzBlue;
};

// bitmap file header
struct BITMAPFILEHEADER
{
	unsigned short bfType;
	unsigned int bfSize;
	unsigned short bfReserved1;
	unsigned short bfReserved2;
	unsigned int bfOffBits;
};

// bitmap info header
struct BITMAPINFOHEADER
{
	unsigned int biSize;
	unsigned int biWidth;
	unsigned int biHeight;
	unsigned short biPlanes;
	unsigned short biBitCount;
	unsigned int biCompression;
	unsigned int biSizeImage;
	unsigned int biXPelsPerMeter;
	unsigned int biYPelsPerMeter;
	unsigned int biClrUsed;
	unsigned int biClrImportant;
	unsigned int biRedMask;
	unsigned int biGreenMask;
	unsigned int biBlueMask;
	unsigned int biAlphaMask;
	unsigned int biCSType;
	CIEXYZTRIPLE biEndpoints;
	unsigned int biGammaRed;
	unsigned int biGammaGreen;
	unsigned int biGammaBlue;
	unsigned int biIntent;
	unsigned int biProfileData;
	unsigned int biProfileSize;
	unsigned int biReserved;
};

// rgb quad
struct RGBQUAD
{
	unsigned char rgbBlue;
	unsigned char rgbGreen;
	unsigned char rgbRed;
	unsigned char rgbReserved;
};

// read bytes
template <typename Type>
void read(std::ifstream &fp, Type &result, std::size_t size)
{
	fp.read(reinterpret_cast<char *>(&result), size);
}

template <typename Type>
void write(std::ofstream &fp, Type par, std::size_t size)
{
	fp.write(reinterpret_cast<char *>(&par), size);
}

unsigned char bitextract(const unsigned int byte, const unsigned int mask);
void readBMP(int argc, char **argv, int *&rgbInfo, BITMAPINFOHEADER &fileInfoHeader);
void writeBMP(char **argv, int *rgbInfo, BITMAPINFOHEADER fileInfoHeader);

texture<int, 2, hipReadModeElementType> texRef;
__global__ void bilateralBlur(int *filteredImage, int W, int r)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	float sum0 = 0.0f, sum1 = 0.0f, sum2 = 0.0f, sum3 = 0.0f;
	float result0 = 0.0f, result1 = 0.0f, result2 = 0.0f, result3 = 0.0f;
	float c0, c1, c2, c3, cl0, cl1, cl2, cl3;
	int mask0 = 0xff, mask1 = 0xff00, mask2 = 0xff0000, mask3 = 0xff000000;
	int c = tex2D(texRef, idx, idy);

	int clr;
	c0 = (c & mask0) / 255.0f;
	c1 = ((unsigned int)(c & mask1) >> 8) / 255.0f;
	c2 = ((unsigned int)(c & mask2) >> 16) / 255.0f;
	c3 = ((unsigned int)(c & mask3) >> 24) / 255.0f;
	for (int ix = -r; ix <= r; ++ix)
		for (int iy = -r; iy <= r; ++iy)
		{
			clr = tex2D(texRef, idx + ix, idy + iy);
			cl0 = (float)(clr & mask0) / 255.0f;
			cl1 = (float)((unsigned int)(clr & mask1) >> 8) / 255.0f;
			cl2 = (float)((unsigned int)(clr & mask2) >> 16) / 255.0f;
			cl3 = (float)((unsigned int)(clr & mask3) >> 24) / 255.0f;
			float w0 = expf((-SQR(ix) - SQR(iy) - SQR(cl0 - c0)) / SQR(r));
			float w1 = expf((-SQR(ix) - SQR(iy) - SQR(cl1 - c1)) / SQR(r));
			float w2 = expf((-SQR(ix) - SQR(iy) - SQR(cl2 - c2)) / SQR(r));
			float w3 = expf((-SQR(ix) - SQR(iy) - SQR(cl3 - c3)) / SQR(r));
			result0 += w0 * cl0;
			result1 += w1 * cl1;
			result2 += w2 * cl2;
			result3 += w3 * cl3;
			sum0 += w0;
			sum1 += w1;
			sum2 += w2;
			sum3 += w3;
		}
	result0 /= sum0;
	result1 /= sum1;
	result2 /= sum2;
	result3 /= sum3;
	filteredImage[idx + idy * W] = ((unsigned int)(result3 * 255.0f) << 24) + ((unsigned int)(result2 * 255.0f) << 16) + ((unsigned int)(result1 * 255.0f) << 8) + (unsigned int)(result0 * 255.0f);
}

BITMAPFILEHEADER fileHeader;

int main(int argc, char *argv[])
{
	int *rgbInfo, *devPtr;
	BITMAPINFOHEADER fileInfoHeader;
	readBMP(argc, argv, rgbInfo, fileInfoHeader);
	unsigned long int pitch;
	// Здесь у нас имеется одномерный массив цветов пикселей в формате rgb
	// Применить ядро (Возможно, разделить картинку на равные части и в цикле загружать части картинки, и обрабатывать - как целое)
	// Записать новую картинку+
	hipMallocPitch((void **)&devPtr, &pitch, fileInfoHeader.biWidth * sizeof(int), fileInfoHeader.biHeight);
	hipBindTexture2D(nullptr, &texRef, (void *)devPtr, &texRef.channelDesc, fileInfoHeader.biWidth, fileInfoHeader.biHeight, pitch);
	hipMemcpy2D(devPtr, pitch, rgbInfo, fileInfoHeader.biWidth * sizeof(int), fileInfoHeader.biWidth * sizeof(int), fileInfoHeader.biHeight, hipMemcpyHostToDevice);
	//разделение
	

	bilateralBlur<<<dim3(fileInfoHeader.biWidth/30 + 1,fileInfoHeader.biHeight / 30 + 1,1), dim3(30,30,1)>>>(devPtr, pitch / 4, BLUR_PARAM);
	
	hipMemcpy2D(rgbInfo, fileInfoHeader.biWidth * sizeof(int), devPtr, pitch, fileInfoHeader.biWidth * sizeof(int), fileInfoHeader.biHeight, hipMemcpyDefault);

	writeBMP(argv, rgbInfo, fileInfoHeader);

	delete[] rgbInfo;
	hipFree(devPtr);
	return 0;
}








unsigned char bitextract(const unsigned int byte, const unsigned int mask)
{
	if (mask == 0)
	{
		return 0;
	}

	// определение количества нулевых бит справа от маски
	unsigned int
		maskBufer = mask,
		maskPadding = 0;

	while (!(maskBufer & 1))
	{
		maskBufer >>= 1;
		maskPadding++;
	}

	// применение маски и смещение
	return (unsigned int)(byte & mask) >> maskPadding;
}

void writeBMP(char **argv, int *rgbInfo, BITMAPINFOHEADER fileInfoHeader)
{ // полный rgbInfo сюда заносим
	char *fileName = argv[2];
	std::ofstream fileStream;
	fileStream.open(fileName, std::ifstream::binary);
	if (!fileStream)
	{
		std::cout << "Error opening file '" << fileName << "'." << std::endl;
		exit(0);
	}
	write(fileStream, fileHeader.bfType, sizeof(fileHeader.bfType));
	write(fileStream, fileHeader.bfSize, sizeof(fileHeader.bfSize));
	write(fileStream, fileHeader.bfReserved1, sizeof(fileHeader.bfReserved1));
	write(fileStream, fileHeader.bfReserved2, sizeof(fileHeader.bfReserved2));
	write(fileStream, fileHeader.bfOffBits, sizeof(fileHeader.bfOffBits));
	//.................................................

	// информация изображения
	write(fileStream, fileInfoHeader.biSize, sizeof(fileInfoHeader.biSize));

	// bmp core
	if (fileInfoHeader.biSize >= 12)
	{
		write(fileStream, fileInfoHeader.biWidth, sizeof(fileInfoHeader.biWidth));
		write(fileStream, fileInfoHeader.biHeight, sizeof(fileInfoHeader.biHeight));
		write(fileStream, fileInfoHeader.biPlanes, sizeof(fileInfoHeader.biPlanes));
		write(fileStream, fileInfoHeader.biBitCount, sizeof(fileInfoHeader.biBitCount));
	}

	// получаем информацию о битности
	int colorsCount = (unsigned)fileInfoHeader.biBitCount >> 3;
	if (colorsCount < 3)
	{
		colorsCount = 3;
	}

	int bitsOnColor = fileInfoHeader.biBitCount / colorsCount;
	int maskValue = (1 << bitsOnColor) - 1;

	// bmp v1
	if (fileInfoHeader.biSize >= 40)
	{
		write(fileStream, fileInfoHeader.biCompression, sizeof(fileInfoHeader.biCompression));
		write(fileStream, fileInfoHeader.biSizeImage, sizeof(fileInfoHeader.biSizeImage));
		write(fileStream, fileInfoHeader.biXPelsPerMeter, sizeof(fileInfoHeader.biXPelsPerMeter));
		write(fileStream, fileInfoHeader.biYPelsPerMeter, sizeof(fileInfoHeader.biYPelsPerMeter));
		write(fileStream, fileInfoHeader.biClrUsed, sizeof(fileInfoHeader.biClrUsed));
		write(fileStream, fileInfoHeader.biClrImportant, sizeof(fileInfoHeader.biClrImportant));
	}

	// bmp v2
	fileInfoHeader.biRedMask = 0;
	fileInfoHeader.biGreenMask = 0;
	fileInfoHeader.biBlueMask = 0;

	if (fileInfoHeader.biSize >= 52)
	{
		write(fileStream, fileInfoHeader.biRedMask, sizeof(fileInfoHeader.biRedMask));
		write(fileStream, fileInfoHeader.biGreenMask, sizeof(fileInfoHeader.biGreenMask));
		write(fileStream, fileInfoHeader.biBlueMask, sizeof(fileInfoHeader.biBlueMask));
	}

	// если маска не задана, то ставим маску по умолчанию
	if (fileInfoHeader.biRedMask == 0 || fileInfoHeader.biGreenMask == 0 || fileInfoHeader.biBlueMask == 0)
	{
		fileInfoHeader.biRedMask = maskValue << (bitsOnColor * 2);
		fileInfoHeader.biGreenMask = maskValue << bitsOnColor;
		fileInfoHeader.biBlueMask = maskValue;
	}

	// bmp v3
	if (fileInfoHeader.biSize >= 56)
	{
		write(fileStream, fileInfoHeader.biAlphaMask, sizeof(fileInfoHeader.biAlphaMask));
	}
	else
	{
		fileInfoHeader.biAlphaMask = maskValue << (bitsOnColor * 3);
	}

	// bmp v4
	if (fileInfoHeader.biSize >= 108)
	{
		write(fileStream, fileInfoHeader.biCSType, sizeof(fileInfoHeader.biCSType));
		write(fileStream, fileInfoHeader.biEndpoints, sizeof(fileInfoHeader.biEndpoints));
		write(fileStream, fileInfoHeader.biGammaRed, sizeof(fileInfoHeader.biGammaRed));
		write(fileStream, fileInfoHeader.biGammaGreen, sizeof(fileInfoHeader.biGammaGreen));
		write(fileStream, fileInfoHeader.biGammaBlue, sizeof(fileInfoHeader.biGammaBlue));
	}

	// bmp v5
	if (fileInfoHeader.biSize >= 124)
	{
		write(fileStream, fileInfoHeader.biIntent, sizeof(fileInfoHeader.biIntent));
		write(fileStream, fileInfoHeader.biProfileData, sizeof(fileInfoHeader.biProfileData));
		write(fileStream, fileInfoHeader.biProfileSize, sizeof(fileInfoHeader.biProfileSize));
		write(fileStream, fileInfoHeader.biReserved, sizeof(fileInfoHeader.biReserved));
	}
	int linePadding = ((fileInfoHeader.biWidth * (fileInfoHeader.biBitCount / 8)) % 4) & 3;
	for (unsigned int i = 0; i < fileInfoHeader.biHeight; i++)
	{
		for (unsigned int j = 0; j < fileInfoHeader.biWidth; j++)
		{
			write(fileStream, rgbInfo[i * fileInfoHeader.biWidth + j], 3);
		}
		for (int j = 0; j < linePadding; j++)
			write(fileStream, 0, sizeof(int));
	}
}

void readBMP(int argc, char **argv, int *&rgbInfo, BITMAPINFOHEADER &fileInfoHeader)
{
	unsigned char rgbRed, rgbGreen, rgbBlue, rgbReserved;
	if (argc < 2)
	{
		std::cout << "Usage: " << argv[0] << " file_name" << std::endl;
		exit(0);
	}

	char *fileName = argv[1];

	// открываем файл
	std::ifstream fileStream(fileName, std::ifstream::binary);
	if (!fileStream)
	{
		std::cout << "Error opening file '" << fileName << "'." << std::endl;
		exit(0);
	}

	// заголовок изображения
	read(fileStream, fileHeader.bfType, sizeof(fileHeader.bfType));
	read(fileStream, fileHeader.bfSize, sizeof(fileHeader.bfSize));
	read(fileStream, fileHeader.bfReserved1, sizeof(fileHeader.bfReserved1));
	read(fileStream, fileHeader.bfReserved2, sizeof(fileHeader.bfReserved2));
	read(fileStream, fileHeader.bfOffBits, sizeof(fileHeader.bfOffBits));

	if (fileHeader.bfType != 0x4D42)
	{
		std::cout << "Error: '" << fileName << "' is not BMP file." << std::endl;
		exit(0);
	}

	// информация изображения
	read(fileStream, fileInfoHeader.biSize, sizeof(fileInfoHeader.biSize));

	// bmp core
	if (fileInfoHeader.biSize >= 12)
	{
		read(fileStream, fileInfoHeader.biWidth, sizeof(fileInfoHeader.biWidth));
		read(fileStream, fileInfoHeader.biHeight, sizeof(fileInfoHeader.biHeight));
		read(fileStream, fileInfoHeader.biPlanes, sizeof(fileInfoHeader.biPlanes));
		read(fileStream, fileInfoHeader.biBitCount, sizeof(fileInfoHeader.biBitCount));
	}

	// получаем информацию о битности
	int colorsCount = fileInfoHeader.biBitCount >> 3;
	if (colorsCount < 3)
	{
		colorsCount = 3;
	}

	int bitsOnColor = fileInfoHeader.biBitCount / colorsCount;
	int maskValue = (1 << bitsOnColor) - 1;

	if (fileInfoHeader.biSize >= 40)
	{
		read(fileStream, fileInfoHeader.biCompression, sizeof(fileInfoHeader.biCompression));
		read(fileStream, fileInfoHeader.biSizeImage, sizeof(fileInfoHeader.biSizeImage));
		read(fileStream, fileInfoHeader.biXPelsPerMeter, sizeof(fileInfoHeader.biXPelsPerMeter));
		read(fileStream, fileInfoHeader.biYPelsPerMeter, sizeof(fileInfoHeader.biYPelsPerMeter));
		read(fileStream, fileInfoHeader.biClrUsed, sizeof(fileInfoHeader.biClrUsed));
		read(fileStream, fileInfoHeader.biClrImportant, sizeof(fileInfoHeader.biClrImportant));
	}

	// bmp v2
	fileInfoHeader.biRedMask = 0;
	fileInfoHeader.biGreenMask = 0;
	fileInfoHeader.biBlueMask = 0;

	if (fileInfoHeader.biSize >= 52)
	{
		read(fileStream, fileInfoHeader.biRedMask, sizeof(fileInfoHeader.biRedMask));
		read(fileStream, fileInfoHeader.biGreenMask, sizeof(fileInfoHeader.biGreenMask));
		read(fileStream, fileInfoHeader.biBlueMask, sizeof(fileInfoHeader.biBlueMask));
	}

	// если маска не задана, то ставим маску по умолчанию
	if (fileInfoHeader.biRedMask == 0 || fileInfoHeader.biGreenMask == 0 || fileInfoHeader.biBlueMask == 0)
	{
		fileInfoHeader.biRedMask = maskValue << (bitsOnColor * 2);
		fileInfoHeader.biGreenMask = maskValue << bitsOnColor;
		fileInfoHeader.biBlueMask = maskValue;
	}

	// bmp v3
	if (fileInfoHeader.biSize >= 56)
	{
		read(fileStream, fileInfoHeader.biAlphaMask, sizeof(fileInfoHeader.biAlphaMask));
	}
	else
	{
		fileInfoHeader.biAlphaMask = maskValue << (bitsOnColor * 3);
	}

	// bmp v4
	if (fileInfoHeader.biSize >= 108)
	{
		read(fileStream, fileInfoHeader.biCSType, sizeof(fileInfoHeader.biCSType));
		read(fileStream, fileInfoHeader.biEndpoints, sizeof(fileInfoHeader.biEndpoints));
		read(fileStream, fileInfoHeader.biGammaRed, sizeof(fileInfoHeader.biGammaRed));
		read(fileStream, fileInfoHeader.biGammaGreen, sizeof(fileInfoHeader.biGammaGreen));
		read(fileStream, fileInfoHeader.biGammaBlue, sizeof(fileInfoHeader.biGammaBlue));
	}

	// bmp v5
	if (fileInfoHeader.biSize >= 124)
	{
		read(fileStream, fileInfoHeader.biIntent, sizeof(fileInfoHeader.biIntent));
		read(fileStream, fileInfoHeader.biProfileData, sizeof(fileInfoHeader.biProfileData));
		read(fileStream, fileInfoHeader.biProfileSize, sizeof(fileInfoHeader.biProfileSize));
		read(fileStream, fileInfoHeader.biReserved, sizeof(fileInfoHeader.biReserved));
	}

	// проверка на поддерку этой версии формата
	if (fileInfoHeader.biSize != 12 && fileInfoHeader.biSize != 40 && fileInfoHeader.biSize != 52 &&
		fileInfoHeader.biSize != 56 && fileInfoHeader.biSize != 108 && fileInfoHeader.biSize != 124)
	{
		std::cout << "Error: Unsupported BMP format." << std::endl;
		exit(0);
	}

	if (fileInfoHeader.biBitCount != 16 && fileInfoHeader.biBitCount != 24 && fileInfoHeader.biBitCount != 32)
	{
		std::cout << "Error: Unsupported BMP bit count." << std::endl;
		exit(0);
	}

	if (fileInfoHeader.biCompression != 0 && fileInfoHeader.biCompression != 3)
	{
		std::cout << "Error: Unsupported BMP compression." << std::endl;
		exit(0);
	}

	// rgb info
	rgbInfo = new int[fileInfoHeader.biHeight * fileInfoHeader.biWidth];

	// определение размера отступа в конце каждой строки
	int linePadding = ((fileInfoHeader.biWidth * (fileInfoHeader.biBitCount / 8)) % 4) & 3;

	// чтение
	unsigned int bufer;
	for (unsigned int i = 0; i < fileInfoHeader.biHeight; i++)
	{
		for (unsigned int j = 0; j < fileInfoHeader.biWidth; j++)
		{
			read(fileStream, bufer, fileInfoHeader.biBitCount / 8);
			rgbRed = bitextract(bufer, fileInfoHeader.biRedMask);
			rgbGreen = bitextract(bufer, fileInfoHeader.biGreenMask);
			rgbBlue = bitextract(bufer, fileInfoHeader.biBlueMask);
			rgbReserved = bitextract(bufer, fileInfoHeader.biAlphaMask);
			rgbInfo[i * fileInfoHeader.biWidth + j] = (rgbReserved << 24) + (rgbRed << 16) + (rgbGreen << 8) + rgbBlue;
		}
		fileStream.seekg(linePadding, std::ios_base::cur);
	}
}
